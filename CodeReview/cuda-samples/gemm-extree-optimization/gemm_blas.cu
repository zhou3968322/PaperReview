#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>




int main() {
    const int M = 1024;
    const int N = 1024;
    const int K = 1024;

    // Allocate and initialize host memory
    float *h_A = (float*)malloc(M * K * sizeof(float));
    float *h_B = (float*)malloc(K * N * sizeof(float));
    float *h_C_cublas = (float*)malloc(M * N * sizeof(float));
    float *h_C_custom = (float*)malloc(M * N * sizeof(float));

    // Initialize matrices A and B
    for (int i = 0; i < M * K; i++) h_A[i] = static_cast<float>(rand()) / RAND_MAX;
    for (int i = 0; i < K * N; i++) h_B[i] = static_cast<float>(rand()) / RAND_MAX;

    // Allocate device memory
    float *d_A, *d_B, *d_C_cublas, *d_C_custom;
    hipMalloc(&d_A, M * K * sizeof(float));
    hipMalloc(&d_B, K * N * sizeof(float));
    hipMalloc(&d_C_cublas, M * N * sizeof(float));
    hipMalloc(&d_C_custom, M * N * sizeof(float));

    // Copy data to device
    hipMemcpy(d_A, h_A, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, K * N * sizeof(float), hipMemcpyHostToDevice);

    // cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Timing variables
    hipEvent_t start, stop;
    float milliseconds = 0;

    // cuBLAS GEMM timing
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    cublas_gemm(handle, d_A, d_B, d_C_cublas, M, N, K);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "cuBLAS GEMM Time: " << milliseconds << " ms" << std::endl;

    // Custom GEMM timing
    hipEventRecord(start);

    custom_gemm(d_A, d_B, d_C_custom, M, N, K);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Custom GEMM Time: " << milliseconds << " ms" << std::endl;

    // Cleanup
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C_cublas);
    hipFree(d_C_custom);
    free(h_A);
    free(h_B);
    free(h_C_cublas);
    free(h_C_custom);

    hipblasDestroy(handle);
    return 0;